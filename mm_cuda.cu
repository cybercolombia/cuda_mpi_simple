#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "mm_cuda.h"

__global__ void matrixMultiplyKernel(int *A, int *B, int *C, int rowsToProcess) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < rowsToProcess && col < N) {
    int sum = 0;
    for (int k = 0; k < N; ++k) {
      sum += A[row * N + k] * B[k * N + col];
    }
    C[row * N + col] = sum;
  }
}

void matrixMultiply(int *A, int *B, int *C, int rowsToProcess) {
  int *d_A, *d_B, *d_C;

  // Allocate device memory
  hipMalloc((void **)&d_A, rowsToProcess * N * sizeof(int));
  hipMalloc((void **)&d_B, N * N * sizeof(int));
  hipMalloc((void **)&d_C, rowsToProcess * N * sizeof(int));

  // Copy data to device
  hipMemcpy(d_A, A, rowsToProcess * N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, N * N * sizeof(int), hipMemcpyHostToDevice);

  // Define block and grid sizes
  dim3 threadsPerBlock(16, 16);
  dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                     (rowsToProcess + threadsPerBlock.y - 1) /
                         threadsPerBlock.y);

  // Launch kernel
  matrixMultiplyKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C,
                                                           rowsToProcess);

  // Copy result back to host
  hipMemcpy(C, d_C, rowsToProcess * N * sizeof(int), hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}
